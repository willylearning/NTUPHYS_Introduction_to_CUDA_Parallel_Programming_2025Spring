// Vector Trace (Sum of Elements)
// compile with the following command:
//
//
// (for GTX1060)
// nvcc -arch=compute_61 -code=sm_61,sm_61 -O3 -m64 -o vecAdd vecAdd.cu


// Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Variables
float* h_A;   // host vector
float* h_C;   // host result (partial sums from GPU)
float* d_A;   // device vector
float* d_C;   // device result (partial sums)

// Functions
void RandomInit(float*, int);

// Device code
__global__ void VecTrace(const float* A, float* C, int N)
{
    extern __shared__ float cache[];   //  its size is allocated at runtime call

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0;  // register for each thread
    while (i < N) {
        temp += A[i];
        i += blockDim.x * gridDim.x;   // go to the next grid 
    }
   
    cache[cacheIndex] = temp;   // set the cache value 

    __syncthreads();

    // perform parallel reduction, threadsPerBlock must be 2^m
    int ib = blockDim.x / 2;
    while (ib != 0) {
      if (cacheIndex < ib)
      	cache[cacheIndex] += cache[cacheIndex + ib]; 

      __syncthreads();
      ib /= 2;
    }
    
    if(cacheIndex == 0)
      C[blockIdx.x] = cache[0];

}

// Host code
int main(void)
{
    int gid;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    printf("Enter the GPU ID: ");
    scanf("%d", &gid);
    printf("%d\n", gid);
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Set GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("Vector Trace (Sum of Elements)\n");
    int N = 81920007;   // Fixed array size
    printf("Size of the vector: %d\n", N);

    // set the sizes of threads and blocks
    int threadsPerBlock;
    printf("Enter the number (2^m) of threads per block: ");
    scanf("%d",&threadsPerBlock);
    printf("%d\n",threadsPerBlock);
    if (threadsPerBlock > 1024) {
        printf("The number of threads per block must be less than 1024!\n");
        exit(0);
    }

    int blocksPerGrid;
    printf("Enter the number of blocks per grid: ");
    scanf("%d",&blocksPerGrid);
    // blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("%d\n",blocksPerGrid);
    if (blocksPerGrid > 2147483647) {
        printf("The number of blocks must be less than 2147483647!\n");
        exit(0);
    }

    // allocate input vector h_A and result vector h_C in host memory
    int size = N * sizeof(float);
    int sb = blocksPerGrid * sizeof(float);

    h_A = (float*)malloc(size);
    h_C = (float*)malloc(sb);

    // initialize input vector
    RandomInit(h_A, N);

    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord(start,0);

    // allocate vectors in device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_C, sb);

    // copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // stop the timer for input
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float Intime;
    hipEventElapsedTime(&Intime, start, stop);
    printf("Input time for GPU: %f (ms)\n", Intime);

    // start the timer for computation
    hipEventRecord(start, 0);

    int sm = threadsPerBlock * sizeof(float);
    VecTrace<<<blocksPerGrid, threadsPerBlock, sm>>>(d_A, d_C, N);

    // stop the timer for computation
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float gputime;
    hipEventElapsedTime(&gputime, start, stop);
    printf("Processing time for GPU: %f (ms)\n", gputime);
    printf("GPU Gflops: %f\n", N / (1000000.0 * gputime));

    // start the timer for output
    hipEventRecord(start, 0);

    // copy result from device to host
    hipMemcpy(h_C, d_C, sb, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);

    double h_G = 0.0;
    for (int i = 0; i < blocksPerGrid; i++)
        h_G += (double)h_C[i];

    // stop the timer for output
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float Outime;
    hipEventElapsedTime(&Outime, start, stop);
    printf("Output time for GPU: %f (ms)\n", Outime);

    float gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms)\n", gputime_tot);

    // start the timer for CPU
    hipEventRecord(start, 0);

    // compute reference solution on CPU
    double h_D = 0.0;
    for (int i = 0; i < N; i++)
        h_D += (double)h_A[i];

    // stop the timer for CPU
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime(&cputime, start, stop);
    printf("Processing time for CPU: %f (ms)\n", cputime);
    printf("CPU Gflops: %f\n", N / (1000000.0 * cputime));
    printf("Speedup of GPU = %f\n", cputime / gputime_tot);

    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check result
    printf("Check result:\n");
    double diff = fabs((h_D - h_G) / h_D);
    printf("|(h_G - h_D)/h_D|=%20.15e\n", diff);
    printf("h_G =%20.15e\n", h_G);
    printf("h_D =%20.15e\n", h_D);
    printf("\n");

    free(h_A);
    free(h_C);

    hipDeviceReset();
}

// Allocates an array with random float entries in (-1,1)
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = 2.0 * rand() / (float)RAND_MAX - 1.0;
//        data[i] = 1.0;   // set all elements to one for checking the code.
}



